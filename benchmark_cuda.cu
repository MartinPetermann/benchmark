#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <chrono>

#include "benchmark_cuda.h"

template <class T> __global__ void test_gpu(T limit)

{
    T    sum = 0;
    bool prime;

    const u_int32_t a = blockIdx.x;
    const u_int32_t b = threadIdx.x;

    for (T i = 2; i < limit; i++) {
        prime = true;
        for (T j = 2; j * j <= i; j++) {
            for (T k = 2; j * k <= i; k++) {
                if (j * k == i) {
                    prime = false;
                    break;
                }
            }
            if (prime == false) {
                break;
            }
        }
        if (prime == true) {
            sum += i;
        }
    }
    sum += (a + b);
}

template <class T> void test_cpu(T limit)

{
    T    sum = 0;
    bool prime;

    for (T i = 2; i < limit; i++) {
        prime = true;
        for (T j = 2; j * j <= i; j++) {
            for (T k = 2; j * k <= i; k++) {
                if (j * k == i) {
                    prime = false;
                    break;
                }
            }
            if (prime == false) {
                break;
            }
        }
        if (prime == true) {
            sum += i;
        }
    }
}

class Timer {
private:
    std::chrono::time_point<std::chrono::system_clock> start_time;

public:
    void   start() { start_time = std::chrono::system_clock::now(); }
    double stop()
    {
        return (double)std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now()
                                                                             - start_time)
                   .count()
               / 1000;
    }
};

#define LIMIT 20000


void perftest(bool gpu, bool integer, int thread_blocks, int threads)
{
    Timer t;
    t.start();
    if (gpu) {
        if (integer) {
            test_gpu<u_int32_t><<<thread_blocks, threads>>>(LIMIT);
        } else {
            test_gpu<float><<<thread_blocks, threads>>>(LIMIT);
        }
        hipDeviceSynchronize();
        printf("GPU %10s (%d thread block(s), each %d thread(s)): %fs\n", (integer ? "u_int32_t" : "float"), thread_blocks, threads, t.stop());
    } else {
        if (integer) {
            test_cpu<u_int32_t>(LIMIT);
        } else {
            test_cpu<float>(LIMIT);
        }
        printf("CPU %10s:                                       %fs\n", (integer ? "u_int32_t" : "float"), t.stop());
    }
}


